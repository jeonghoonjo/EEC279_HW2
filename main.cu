#include <utility>
#include <cassert>
#include <iostream>
#include "matrix.hpp"

#define CUDA_CHECK_ERROR(X) do { \
    auto ret_code  = X; \
    if (ret_code != 0) { \
        std::cerr << __FILE__ << ":" << __LINE__ << \
        ": CUDA assertion failed: " << hipGetErrorString(ret_code) << '\n' ; \
        std::flush(std::cout); \
        std::flush(std::cerr); \
        assert(ret_code == 0); \
    } \
} while(0)

void print_matrix(const float* data, size_t rows, size_t cols) {
    std::cout << rows << " x " << cols << '\n';
    for (size_t i = 0; i < rows; ++i) {
        for (size_t j = 0; j < cols; ++j) {
            // print 6 decimal places, padded to 10 total characters
            // printf("% 10.6f   ", mat.data()[mat.rows * i + j]);

            // print 0 decimal places, padded to 5 characters - for wikipedia test data
            printf("%5.0f", data[(cols * i) + j]);
        }
        printf("\n");
    }
    printf("\n");
}

void print_matrix(const matrix::matrix_t& mat) {
    print_matrix(mat.data(), mat.rows, mat.cols);
}

void scale_matrix(
    const float scale,
    const float *b,
    float *c,
    size_t x, // block inx in mat c
    size_t y, // block inx in mat c
    size_t stride, // elements to jump in c to access only the blocks
    size_t b_rows,
    size_t b_cols) {

    for (size_t i = 0; i < b_rows; ++i) {
        for (size_t j = 0; j < b_cols; ++j) {
            auto c_flat_inx = (i + x * b_rows) * stride + (j + y * b_cols);
            c[c_flat_inx] = scale * b[i * b_cols + j];
        }
    }
}

void
unpartitionedKhatriRaoProduct(float * C, const float * A, const float * B,
                              unsigned int ah, unsigned int aw,
                              unsigned int bh, unsigned int bw) {
    for (int i = 0; i < ah; ++i) {
        for (int j = 0; j < aw; ++j) {
            // for each block, scale matrix B with element from A, and store into C
            scale_matrix(
                A[aw * i + j],
                B,
                C,
                i, // locate block a[i,j] * B
                j, // locate block a[i,j] * B
                aw * bw,  // stride is a_col * b_col
                bh,
                bw
            );
        }
    }
}

int main(int argc, const char *argv[]) {

    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " mata matb\n";
        return -1;
    }

    // read in the matrix, returns types matrix::matrix_t, in a std::vector<float>
    const auto mat_a = matrix::load_matrix(std::string { argv[1] });
    const auto mat_b = matrix::load_matrix(std::string { argv[2] });

    // The Khatri-Rao product of two arbitrary-sized unpartitioned matrices. This is
    // the equivalent of the Kronecker product of these two matrices.
    // if A is an m x n matrix and B is a p x q matrix, then the Kronecker product C = A ⊗ B is the
    // pm x qn block matrix
    auto [m, n] = std::pair(mat_a.rows, mat_a.cols); // same as m = mat_a.row, n = mat_a.cols
    auto [p, q] = std::pair(mat_b.rows, mat_b.cols); // same as p = mat_b.row, q = mat_b.cols

    matrix::matrix_t mat_c {};
    mat_c.rows = (m * p);
    mat_c.cols = (n * q);

    // allocate and initialize memory for mat_c
    mat_c.arr.resize( (m * p) * (n * q) );

    unpartitionedKhatriRaoProduct(
        mat_c.data(),
        mat_a.data(),
        mat_b.data(),
        mat_a.rows, mat_a.cols,
        mat_b.rows, mat_b.cols
    );

    print_matrix(mat_a);
    print_matrix(mat_b);
    print_matrix(mat_c);

    return EXIT_SUCCESS;
}
